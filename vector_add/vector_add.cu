#include <hip/hip_runtime.h>
#include <iostream>

__global__ void vecAdd(const float* A, const float* B, float* C, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        C[idx] = A[idx] + B[idx];
    }
}

int main() {
    int N = 1 << 20;                     
    size_t size = N * sizeof(float);

    float *h_A = new float[N], *h_B = new float[N], *h_C = new float[N];
    for (int i = 0; i < N; ++i) { h_A[i] = i; h_B[i] = 2*i; }

    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size); hipMalloc(&d_B, size); hipMalloc(&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    int threads = 256;
    int blocks = (N + threads - 1) / threads;

    hipEvent_t start, stop;
    hipEventCreate(&start); hipEventCreate(&stop);
    hipEventRecord(start);

    vecAdd<<<blocks, threads>>>(d_A, d_B, d_C, N);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float ms;
    hipEventElapsedTime(&ms, start, stop);

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    std::cout << "VectorAdd ("<< N <<" elems) took " << ms << " ms\n";

    hipFree(d_A); hipFree(d_B); hipFree(d_C);
    delete[] h_A; delete[] h_B; delete[] h_C;
    return 0;
}
